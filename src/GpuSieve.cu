#include <cassert>
#include <cmath>
#include <cstring>
#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

const uint64_t MAX_STRIDE = 256000000;
int BLOCK_SIZE = 512;
bool* isPrimeListHost = nullptr;

std::vector<uint64_t> sieveCpuPrep(uint64_t maxNumber) {
    std::vector<bool> isPrimeList;
    std::vector<uint64_t> prepedPrimes;
    std::cout << "joooo" << std::endl;
    isPrimeList.assign(maxNumber, true);

    int sqrtMaxNumber =maxNumber;

    for (uint64_t i = 2; i < sqrtMaxNumber; i++)
    {
        if (isPrimeList[i]) {
            prepedPrimes.push_back(i);

            for (uint64_t j = 2; i * j < maxNumber; j++) {
                isPrimeList[i * j] = false;
            }
        }
    }
    std::cout << "there" << std::endl;
    for (uint64_t i = sqrtMaxNumber; i < maxNumber; i++)
    {
        if (isPrimeList[i]) {
            prepedPrimes.push_back(i);
        }
    }
    return prepedPrimes;
}


__global__ void gpuSieveKernel(uint64_t maxNumber, bool* isPrimeList, uint64_t* prepedPrimes, uint64_t sizeOfPrepedPrimes) {
    uint64_t threadIndex = blockIdx.x * blockDim.x + threadIdx.x; // index = range of 0 up to MAX_STRIDE
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = threadIndex; i <= maxNumber; i += stride) 
    {
        if (i < 2) continue;

        for (uint64_t j = 0; j < sizeOfPrepedPrimes; j++) 
        {
            uint64_t currentNumber = prepedPrimes[j] * i;
            if (currentNumber > maxNumber) break;

            isPrimeList[currentNumber] = false;
        }
    }
}

void gpuSieve(uint64_t maxNumber, std::vector<uint64_t> prepedPrimes) {
    
    bool* isPrimeListDevice = nullptr;
    uint64_t* prepedPrimesDevice = nullptr;

    isPrimeListHost = (bool*)malloc(maxNumber * sizeof(bool));
    hipMalloc(&isPrimeListDevice, maxNumber * sizeof(bool));
    std::memset(isPrimeListHost, true, maxNumber * sizeof(bool));
    hipMemcpy(isPrimeListDevice, isPrimeListHost, maxNumber * sizeof(bool), hipMemcpyHostToDevice);

    uint64_t prepedPrimesSize = prepedPrimes.size();
    uint64_t* prepedPrimesHost = (uint64_t*)malloc(prepedPrimesSize * sizeof(uint64_t));
    memcpy(prepedPrimesHost, prepedPrimes.data(), prepedPrimesSize * sizeof(uint64_t));
    hipMalloc(&prepedPrimesDevice, prepedPrimesSize * sizeof(uint64_t));
    hipMemcpy(prepedPrimesDevice, prepedPrimesHost, prepedPrimesSize * sizeof(uint64_t), hipMemcpyHostToDevice);

    uint64_t numberOfBlocks = (maxNumber + BLOCK_SIZE - 1) / BLOCK_SIZE;
    uint64_t stride = BLOCK_SIZE * numberOfBlocks;

    if (stride > MAX_STRIDE) {
        numberOfBlocks = MAX_STRIDE / BLOCK_SIZE;
    }
    std::cout << "tutaj" << std::endl;
    gpuSieveKernel<<<numberOfBlocks, BLOCK_SIZE >>> (maxNumber, isPrimeListDevice, prepedPrimesDevice, prepedPrimesSize);

    hipDeviceSynchronize();
    hipMemcpy(isPrimeListHost, isPrimeListDevice, maxNumber * sizeof(bool), hipMemcpyDeviceToHost);

    free(prepedPrimesHost);
    hipFree(isPrimeListDevice);
    hipFree(prepedPrimesDevice);
}

void checkPrimescount(int target) {
    uint64_t primesCount = 0;
    for (uint64_t i = 2; i <= target; i++) {
        if (isPrimeListHost[i]) primesCount++;
    }

    std::cout << "Prime numbers: " << primesCount << std::endl;
}




int main() {
    uint64_t target = 1000000000;

    auto startTime = std::chrono::high_resolution_clock::now();

    std::vector<uint64_t> prepedPrimes = sieveCpuPrep(std::sqrt(target));
    gpuSieve(target, prepedPrimes);

    auto endTime = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(endTime - startTime);

    std::cout << "Time taken by threads: "
        << duration.count() << " microseconds" << std::endl;

    checkPrimescount(target);
    return 0;
}
